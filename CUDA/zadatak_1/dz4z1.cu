
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NUMBER_OF_THREADS_PER_BLOCK 256

typedef struct Result
{
    int arithmetic_count;
    int composite_count;
    int n;
    float execution_time;
} Result;

__device__ __host__ void divisor_count_and_sum(unsigned int n, unsigned int *pcount, unsigned int *psum)
{
    unsigned int divisor_count = 1;
    unsigned int divisor_sum = 1;
    unsigned int power = 2;
    for (; (n & 1) == 0; power <<= 1, n >>= 1)
    {
        ++divisor_count;
        divisor_sum += power;
    }
    for (unsigned int p = 3; p * p <= n; p += 2)
    {
        unsigned int count = 1, sum = 1;
        for (power = p; n % p == 0; power *= p, n /= p)
        {
            ++count;
            sum += power;
        }
        divisor_count *= count;
        divisor_sum *= sum;
    }
    if (n > 1)
    {
        divisor_count *= 2;
        divisor_sum *= n + 1;
    }
    *pcount = divisor_count;
    *psum = divisor_sum;
}

__global__ void findArithmeticNumbersKernel(unsigned int *arithmetic_count_gpu, unsigned int *composite_count_gpu, int start, int number_of_iterations)
{
    __shared__ unsigned int counters[2];

    counters[0] = 0;
    counters[1] = 0;

    if (blockIdx.x * blockDim.x + threadIdx.x < number_of_iterations)
    {
        unsigned int divisor_count;
        unsigned int divisor_sum;
        unsigned int myNumber = start + (blockIdx.x * blockDim.x + threadIdx.x);
        divisor_count_and_sum(myNumber, &divisor_count, &divisor_sum);
        if (divisor_sum % divisor_count == 0)
        {
            atomicAdd(&counters[0], 1);
            if (divisor_count > 2)
            {
                atomicAdd(&counters[1], 1);
            }
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        atomicAdd(arithmetic_count_gpu, counters[0]);
        atomicAdd(composite_count_gpu, counters[1]);
    }
}

Result *arithmeticNumbersCPU(char **argv)
{
    Result *result = (Result *)malloc(sizeof(Result));

    int num = atoi(argv[1]);
    unsigned int arithmetic_count = 0;
    unsigned int composite_count = 0;
    unsigned int n;

    struct timespec start_time, end_time;

    clock_gettime(CLOCK_MONOTONIC, &start_time);

    for (n = 1; arithmetic_count <= num; ++n)
    {
        unsigned int divisor_count;
        unsigned int divisor_sum;
        divisor_count_and_sum(n, &divisor_count, &divisor_sum);
        if (divisor_sum % divisor_count != 0)
            continue;
        ++arithmetic_count;
        if (divisor_count > 2)
            ++composite_count;
    }

    clock_gettime(CLOCK_MONOTONIC, &end_time);

    result->arithmetic_count = arithmetic_count;
    result->composite_count = composite_count;
    result->n = n;
    result->execution_time = (end_time.tv_sec - start_time.tv_sec) + (end_time.tv_nsec - start_time.tv_nsec) / 1e9;

    return result;
}

Result *arithmeticNumbersGPU(char **argv)
{
    Result *result = (Result *)malloc(sizeof(Result));

    int num = atoi(argv[1]);
    unsigned int arithmetic_count_cpu = 0, *arithmetic_count_gpu;
    unsigned int composite_count_cpu = 0, *composite_count_gpu;
    unsigned int n = 1;
    unsigned int start = 1;
    unsigned int number_of_iterations = 1;

    // Dummy call - purpose is to set up CUDA environment here so that initialization overhead isn't included in profiling
    // statistics of actual useful CUDA API calls.
    hipDeviceSynchronize();

    hipEvent_t start_time = hipEvent_t();
    hipEvent_t end_time = hipEvent_t();
    hipEventCreate(&start_time);
    hipEventCreate(&end_time);

    hipEventRecord(start_time, 0);

    hipMalloc(&arithmetic_count_gpu, sizeof(unsigned int));
    hipMalloc(&composite_count_gpu, sizeof(unsigned int));

    hipMemset(arithmetic_count_gpu, 0, sizeof(unsigned int));
    hipMemset(composite_count_gpu, 0, sizeof(unsigned int));

    while (arithmetic_count_cpu <= num)
    {
        number_of_iterations = num + 1 - arithmetic_count_cpu;
        n += number_of_iterations;

        dim3 gridDimension((number_of_iterations + NUMBER_OF_THREADS_PER_BLOCK - 1) / NUMBER_OF_THREADS_PER_BLOCK);
        dim3 blockDimension(NUMBER_OF_THREADS_PER_BLOCK);

        findArithmeticNumbersKernel<<<gridDimension, blockDimension>>>(arithmetic_count_gpu, composite_count_gpu, start, number_of_iterations);

        hipMemcpy(&arithmetic_count_cpu, arithmetic_count_gpu, sizeof(unsigned int), hipMemcpyDeviceToHost);

        start += number_of_iterations;
    }

    hipMemcpy(&composite_count_cpu, composite_count_gpu, sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipEventRecord(end_time, 0);
    hipEventSynchronize(end_time);
    float execution_time;
    hipEventElapsedTime(&execution_time, start_time, end_time);

    hipEventDestroy(start_time);
    hipEventDestroy(end_time);

    result->arithmetic_count = arithmetic_count_cpu;
    result->composite_count = composite_count_cpu;
    result->n = n;
    result->execution_time = execution_time / 1000;

    return result;
}

int are_results_equal(Result *sequential_result, Result *parallel_result)
{
    if (sequential_result->arithmetic_count == parallel_result->arithmetic_count &&
        sequential_result->composite_count == parallel_result->composite_count &&
        sequential_result->n == parallel_result->n)
        return 1;
    return 0;
}

int main(int argc, char **argv)
{
    Result *sequential_result, *parallel_result;

    sequential_result = arithmeticNumbersCPU(argv);
    parallel_result = arithmeticNumbersGPU(argv);

    printf("Sequential implementation execution time: %fs\n", sequential_result->execution_time);
    printf("Parallel implementation execution time: %fs\n", parallel_result->execution_time);
    if (are_results_equal(sequential_result, parallel_result))
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");

    free(sequential_result);
    free(parallel_result);

    return 0;
}